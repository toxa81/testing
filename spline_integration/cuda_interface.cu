#include "hip/hip_runtime.h"
// This file must be compiled with nvcc
#include "cuda_interface.h"

extern "C" void print_cuda_timers()
{
    CUDA_timer::cuda_timers_wrapper().print();
}

//================
// CUDA functions
//================

hipStream_t* streams;

extern "C" {

void cuda_initialize()
{
    //CALL_CUDA(hipSetDeviceFlags, (hipDeviceMapHost));
}

void* cuda_malloc(size_t size)
{
    void* ptr;
    CALL_CUDA(hipMalloc, (&ptr, size));
    return ptr;
}

void cuda_free(void* ptr)
{
    CALL_CUDA(hipFree, (ptr));
}

void* cuda_malloc_host(size_t size)
{
    void* ptr;
    CALL_CUDA(hipHostMalloc, (&ptr, size));
    return ptr;
}

void cuda_free_host(void* ptr)
{
    CALL_CUDA(hipHostFree, (ptr));
}

void cuda_copy_to_device(void* target, void const* source, size_t size)
{
    CALL_CUDA(hipMemcpy, (target, source, size, hipMemcpyHostToDevice));
}

void cuda_copy_to_host(void* target, void const* source, size_t size)
{
    CALL_CUDA(hipMemcpy, (target, source, size, hipMemcpyDeviceToHost));
}

void cuda_copy_device_to_device(void* target, void const* source, size_t size)
{
    CALL_CUDA(hipMemcpy, (target, source, size, hipMemcpyDeviceToDevice));
}

void cuda_device_synchronize()
{
    CALL_CUDA(hipDeviceSynchronize, ());
}

void cuda_device_reset()
{
    CALL_CUDA(hipDeviceReset, ());
}

void cuda_create_streams(int num_streams)
{
    streams = (hipStream_t*)malloc(num_streams * sizeof(hipStream_t));
    //for (int i = 0; i < num_streams; i++) hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    for (int i = 0; i < num_streams; i++)
    {
        CALL_CUDA(hipStreamCreate, (&streams[i]));
    }
}

void cuda_destroy_streams(int num_streams)
{
    for (int i = 0; i < num_streams; i++) 
    {
        CALL_CUDA(hipStreamDestroy, (streams[i]));
    }
    free(streams);
}

void cuda_stream_synchronize(int stream_id)
{
    CALL_CUDA(hipStreamSynchronize, (streams[stream_id]));
}

void cuda_async_copy_to_device(void* target, void* source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    CALL_CUDA(hipMemcpyAsync, (target, source, size, hipMemcpyHostToDevice, stream));
}

void cuda_async_copy_to_host(void* target, void* source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    CALL_CUDA(hipMemcpyAsync, (target, source, size, hipMemcpyDeviceToHost, stream));
}

void cuda_async_copy_device_to_device(void* target, void const* source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    CALL_CUDA(hipMemcpyAsync, (target, source, size, hipMemcpyDeviceToDevice, stream));
}

void cuda_memset(void* ptr, int value, size_t size)
{
    CALL_CUDA(hipMemset, (ptr, value, size));
}

void cuda_host_register(void* ptr, size_t size)
{
    assert(ptr);
    
    CALL_CUDA(hipHostRegister, (ptr, size, hipHostRegisterMapped));
}

void cuda_host_unregister(void* ptr)
{
    CALL_CUDA(hipHostUnregister, (ptr));
}

size_t cuda_get_free_mem()
{
    size_t free, total;
    CALL_CUDA(hipMemGetInfo, (&free, &total));

    return free;
}

void cuda_device_info()
{
    int count;
    CALL_CUDA(hipGetDeviceCount, (&count));

    if (count == 0)
    {
        printf("CUDA devices not found\n");
        exit(-100);
    }

    hipDeviceProp_t devprop;
     
    CALL_CUDA(hipGetDeviceProperties, (&devprop, 0));
    
    printf("name                        : %s \n", devprop.name);
    printf("major                       : %i \n", devprop.major);
    printf("minor                       : %i \n", devprop.minor);
    printf("asyncEngineCount            : %i \n", devprop.asyncEngineCount);
    printf("canMapHostMemory            : %i \n", devprop.canMapHostMemory);
    printf("clockRate                   : %i kHz \n", devprop.clockRate);
    printf("concurrentKernels           : %i \n", devprop.concurrentKernels);
    printf("ECCEnabled                  : %i \n", devprop.ECCEnabled);
    printf("l2CacheSize                 : %i kB \n", devprop.l2CacheSize/1024);
    printf("maxGridSize                 : %i %i %i \n", devprop.maxGridSize[0], devprop.maxGridSize[1], devprop.maxGridSize[2]);
    printf("maxThreadsDim               : %i %i %i \n", devprop.maxThreadsDim[0], devprop.maxThreadsDim[1], devprop.maxThreadsDim[2]);
    printf("maxThreadsPerBlock          : %i \n", devprop.maxThreadsPerBlock);
    printf("maxThreadsPerMultiProcessor : %i \n", devprop.maxThreadsPerMultiProcessor);
    printf("memoryBusWidth              : %i bits \n", devprop.memoryBusWidth);
    printf("memoryClockRate             : %i kHz \n", devprop.memoryClockRate);
    printf("memPitch                    : %zi \n", devprop.memPitch);
    printf("multiProcessorCount         : %i \n", devprop.multiProcessorCount);
    printf("regsPerBlock                : %i \n", devprop.regsPerBlock);
    printf("sharedMemPerBlock           : %li kB \n", devprop.sharedMemPerBlock/1024);
    printf("totalConstMem               : %li kB \n", devprop.totalConstMem/1024);
    printf("totalGlobalMem              : %li kB \n", devprop.totalGlobalMem/1024);
    printf("available memory            : %li kB \n", cuda_get_free_mem() / 1024);
}

void cuda_check_last_error()
{
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error != hipSuccess\n");
    }
}

} // extern "C"


void cuda_memcpy2D_device_to_device(void* dst__, size_t ld1__, const void* src__, size_t ld2__, size_t nrow__, size_t ncol__, int elem_size__)
{
    CALL_CUDA(hipMemcpy2D, (dst__, ld1__ * elem_size__, src__, ld2__ * elem_size__, nrow__ * elem_size__, ncol__, hipMemcpyDeviceToDevice));
}


//==================
// CUBLAS functions
//==================

hipblasHandle_t cublas_null_stream_handle;
hipblasHandle_t* cublas_stream_handles;

void cublas_error_message(hipblasStatus_t status)
{
    switch (status)
    {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
        {
            printf("the library was not initialized\n");
            break;
        }
        case HIPBLAS_STATUS_INVALID_VALUE:
        {
            printf("the parameters m,n,k<0\n");
            break;
        }
        case HIPBLAS_STATUS_ARCH_MISMATCH:
        {
            printf("the device does not support double-precision\n");
            break;
        }
        case HIPBLAS_STATUS_EXECUTION_FAILED:
        {
            printf("the function failed to launch on the GPU\n");
            break;
        }
        default:
        {
            printf("cublas status unknown");
        }
    }
}

#define CALL_CUBLAS(func__, args__)                                                 \
{                                                                                   \
    hipblasStatus_t status;                                                          \
    if ((status = func__ args__) != HIPBLAS_STATUS_SUCCESS)                          \
    {                                                                               \
        cublas_error_message(status);                                               \
        char nm[1024];                                                              \
        gethostname(nm, 1024);                                                      \
        printf("hostname: %s\n", nm);                                               \
        printf("Error in %s at line %i of file %s\n", #func__, __LINE__, __FILE__); \
        exit(-100);                                                                 \
    }                                                                               \
}

extern "C" void cublas_create_handles(int num_handles)
{
    CALL_CUBLAS(hipblasCreate, (&cublas_null_stream_handle));
    
    cublas_stream_handles = (hipblasHandle_t*)malloc(num_handles * sizeof(hipblasHandle_t));
    for (int i = 0; i < num_handles; i++)
    {
        CALL_CUBLAS(hipblasCreate, (&cublas_stream_handles[i]));

        CALL_CUBLAS(hipblasSetStream, (cublas_stream_handles[i], streams[i]));
    }
}

extern "C" void cublas_destroy_handles(int num_handles)
{
    CALL_CUBLAS(hipblasDestroy, (cublas_null_stream_handle));
    for (int i = 0; i < num_handles; i++)
    {
        CALL_CUBLAS(hipblasDestroy, (cublas_stream_handles[i]));
    }
}

//== extern "C" void cublas_set_stream(int stream_id__)
//== {
//==     hipStream_t stream = (stream_id__ == -1) ? NULL : streams[stream_id__];
//==     hipblasSetStream(cublas_handle(), stream);
//== }

extern "C" void cublas_zgemv(int transa, int32_t m, int32_t n, hipDoubleComplex* alpha, hipDoubleComplex* a, int32_t lda, 
                             hipDoubleComplex* x, int32_t incx, hipDoubleComplex* beta, hipDoubleComplex* y, int32_t incy, 
                             int stream_id)
{
    const hipblasOperation_t trans[] = {HIPBLAS_OP_N, HIPBLAS_OP_T, HIPBLAS_OP_C};
    hipblasHandle_t handle = (stream_id == -1) ? cublas_null_stream_handle : cublas_stream_handles[stream_id];

    CALL_CUBLAS(hipblasZgemv, (handle, trans[transa], m, n, alpha, a, lda, x, incx, beta, y, incy));
}

extern "C" void cublas_zgemm(int transa, int transb, int32_t m, int32_t n, int32_t k, 
                             hipDoubleComplex* alpha, hipDoubleComplex* a, int32_t lda, hipDoubleComplex* b, 
                             int32_t ldb, hipDoubleComplex* beta, hipDoubleComplex* c, int32_t ldc, int stream_id)
{
    const hipblasOperation_t trans[] = {HIPBLAS_OP_N, HIPBLAS_OP_T, HIPBLAS_OP_C};
    hipblasHandle_t handle = (stream_id == -1) ? cublas_null_stream_handle : cublas_stream_handles[stream_id];
    
    CALL_CUBLAS(hipblasZgemm, (handle, trans[transa], trans[transb], m, n, k, alpha, a, lda, b, ldb, beta, c, ldc));
}

// A(GPU) => B(CPU)
extern "C" void cublas_get_matrix(int rows, int cols, int elemSize, const void *A_device, int lda, void *B_host, int ldb)
{
    CALL_CUBLAS(hipblasGetMatrix, (rows, cols, elemSize, A_device, lda, B_host, ldb));
}

extern "C" void cublas_get_matrix_async(int rows, int cols, int elemSize, const void *A_device, int lda, void *B_host, int ldb, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    CALL_CUBLAS(hipblasGetMatrixAsync, (rows, cols, elemSize, A_device, lda, B_host, ldb, stream));
}

// A(CPU) => B(GPU)
extern "C" void cublas_set_matrix(int rows, int cols, int elemSize, const void *A_host, int lda, void *B_device, int ldb)
{
    CALL_CUBLAS(hipblasSetMatrix, (rows, cols, elemSize, A_host, lda, B_device, ldb));
}

extern "C" void cublas_set_matrix_async(int rows, int cols, int elemSize, const void *A_host, int lda, void *B_device, int ldb, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    CALL_CUBLAS(hipblasSetMatrixAsync, (rows, cols, elemSize, A_host, lda, B_device, ldb, stream));
}

// x(CPU) => y(GPU)
extern "C" void cublas_set_vector(int n, int elemSize, const void *x, int incx, void *y, int incy)
{
    CALL_CUBLAS(hipblasSetVector, (n, elemSize, x, incx, y, incy));
}

//=================
// CUFFT functions
//=================

void cufft_error_message(hipfftResult result)
{
    switch (result)
    {
        case HIPFFT_INVALID_PLAN:
        {
            printf("HIPFFT_INVALID_PLAN\n");
            break;
        }
        case HIPFFT_ALLOC_FAILED:
        {
            printf("HIPFFT_ALLOC_FAILED\n");
            break;
        }
        case HIPFFT_INVALID_VALUE:
        {
            printf("HIPFFT_INVALID_VALUE\n");
            break;
        }
        case HIPFFT_INTERNAL_ERROR:
        {
            printf("HIPFFT_INTERNAL_ERROR\n");
            break;
        }
        case HIPFFT_SETUP_FAILED:
        {
            printf("HIPFFT_SETUP_FAILED\n");
            break;
        }
        case HIPFFT_INVALID_SIZE:
        {
            printf("HIPFFT_INVALID_SIZE\n");
            break;
        }
        default:
        {
            printf("unknown error code %i\n", result);
            break;
        }
    }
}

#define CALL_CUFFT(func__, args__)                                                  \
{                                                                                   \
    hipfftResult result;                                                             \
    if ((result = func__ args__) != HIPFFT_SUCCESS)                                  \
    {                                                                               \
        char nm[1024];                                                              \
        gethostname(nm, 1024);                                                      \
        printf("hostname: %s\n", nm);                                               \
        printf("Error in %s at line %i of file %s: ", #func__, __LINE__, __FILE__); \
        cufft_error_message(result);                                                \
        exit(-100);                                                                 \
    }                                                                               \
}

extern "C" void cufft_create_plan_handle(hipfftHandle* plan)
{
    CALL_CUFFT(hipfftCreate, (plan));
}

extern "C" void cufft_destroy_plan_handle(hipfftHandle plan)
{
    CALL_CUFFT(hipfftDestroy, (plan));
}

/** Get the work size for cuFFT */
extern "C" size_t cufft_get_size(int nx, int ny, int nz, int nfft)
{
    int fft_size = nx * ny * nz;
    int n[] = {nz, ny, nx};
    size_t work_size;

    CALL_CUFFT(hipfftEstimateMany, (3, n, NULL, 1, fft_size, NULL, 1, fft_size, HIPFFT_Z2Z, nfft, &work_size));
    
    return work_size;
}

extern "C" size_t cufft_create_batch_plan(hipfftHandle plan, int nx, int ny, int nz, int nfft)
{
    int fft_size = nx * ny * nz;
    int n[] = {nz, ny, nx};

    CALL_CUFFT(hipfftSetAutoAllocation, (plan, false));
    
    size_t work_size;
    CALL_CUFFT(hipfftMakePlanMany, (plan, 3, n, n, 1, fft_size, n, 1, fft_size, HIPFFT_Z2Z, nfft, &work_size));

    return work_size;
}

extern "C" void cufft_set_work_area(hipfftHandle plan, void* work_area)
{
    CALL_CUFFT(hipfftSetWorkArea, (plan, work_area));
}

__global__ void cufft_batch_load_gpu_kernel
(
    int fft_size, 
    int num_pw_components, 
    int* map, 
    hipDoubleComplex* data, 
    hipDoubleComplex* fft_buffer
)
{
    int i = blockIdx.y;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_pw_components)
    {
        fft_buffer[array2D_offset(map[idx], i, fft_size)] = data[array2D_offset(idx, i, num_pw_components)];
    }
}

extern "C" void cufft_batch_load_gpu(int fft_size,
                                     int num_pw_components, 
                                     int num_fft,
                                     int* map, 
                                     hipDoubleComplex* data, 
                                     hipDoubleComplex* fft_buffer)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_pw_components, grid_t.x), num_fft);
    
    cuda_memset(fft_buffer, 0, fft_size * num_fft * sizeof(hipDoubleComplex));

    cufft_batch_load_gpu_kernel <<<grid_b, grid_t>>>
    (
        fft_size,
        num_pw_components,
        map,
        data, 
        fft_buffer
    );
}

__global__ void cufft_batch_unload_gpu_kernel
(
    int fft_size, 
    int num_pw_components, 
    int* map, 
    hipDoubleComplex* fft_buffer,
    hipDoubleComplex* data,
    double beta
)
{
    int i = blockIdx.y;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_pw_components) 
    {
        data[array2D_offset(idx, i, num_pw_components)] = hipCadd(
            hipCmul(make_hipDoubleComplex(beta, 0), data[array2D_offset(idx, i, num_pw_components)]),
            hipCdiv(fft_buffer[array2D_offset(map[idx], i, fft_size)], make_hipDoubleComplex(double(fft_size), 0)));
    }
}

extern "C" void cufft_batch_unload_gpu(int fft_size,
                                       int num_pw_components,
                                       int num_fft,
                                       int* map, 
                                       hipDoubleComplex* fft_buffer, 
                                       hipDoubleComplex* data,
                                       double beta)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_pw_components, grid_t.x), num_fft);

    cufft_batch_unload_gpu_kernel <<<grid_b, grid_t>>>
    (
        fft_size, 
        num_pw_components, 
        map, 
        fft_buffer,
        data,
        beta
    );
}

extern "C" void cufft_forward_transform(hipfftHandle plan, hipDoubleComplex* fft_buffer)
{
    CUDA_timer t("cufft_forward_transform");
    CALL_CUFFT(hipfftExecZ2Z, (plan, fft_buffer, fft_buffer, HIPFFT_FORWARD));
}

extern "C" void cufft_backward_transform(hipfftHandle plan, hipDoubleComplex* fft_buffer)
{
    CUDA_timer t("cufft_backward_transform");
    CALL_CUFFT(hipfftExecZ2Z, (plan, fft_buffer, fft_buffer, HIPFFT_BACKWARD));
}


__global__ void spline_inner_product_gpu_kernel_v2(int size__, double const* x__, double const* dx__, double const* f__, double const* g__, double* result__)
{
    //int nb = num_blocks(size__, blockDim.x);
    int ib = blockIdx.x;

    extern __shared__ char sdata_ptr[];
    double* sdata = (double*)&sdata_ptr[0];

    int a_offs = 0 * size__;
    int b_offs = 1 * size__;
    int c_offs = 2 * size__;
    int d_offs = 3 * size__;

    sdata[threadIdx.x] = 0;

    //for (int ib = 0; ib < nb; ib++)
    //{
        int i = ib * blockDim.x + threadIdx.x;
        if (i < size__ - 1)
        {
            double xi = x__[i];
            double dxi = dx__[i];

            double a1 = f__[a_offs + i];
            double b1 = f__[b_offs + i];
            double c1 = f__[c_offs + i];
            double d1 = f__[d_offs + i];
            
            double a2 = g__[a_offs + i];
            double b2 = g__[b_offs + i];
            double c2 = g__[c_offs + i];
            double d2 = g__[d_offs + i];
                
            double a1a2 = a1 * a2;
            double d1d2 = d1 * d2;
                
            double k1 = d1 * b2 + c1 * c2 + b1 * d2;

            double k2 = d1 * a2 + c1 * b2 + b1 * c2 + a1 * d2;

            double k3 = c1 * a2 + b1 * b2 + a1 * c2;

            double k4 = d1 * c2 + c1 * d2;
            
            double k5 = b1 * a2 + a1 * b2;

            sdata[threadIdx.x] += dxi * ((a1a2 * xi * xi) + 
                                  dxi * ((xi * (2.0 * a1a2 + xi * k5)) / 2.0 +
                                  dxi * ((a1a2 + xi * (2.0 * k5 + k3 * xi)) / 3.0 + 
                                  dxi * ((k5 + xi * (2.0 * k3 + k2 * xi)) / 4.0 +
                                  dxi * ((k3 + xi * (2.0 * k2 + k1 * xi)) / 5.0 + 
                                  dxi * ((k2 + xi * (2.0 * k1 + k4 * xi)) / 6.0 + 
                                  dxi * ((k1 + xi * (2.0 * k4 + d1d2 * xi)) / 7.0 + 
                                  dxi * ((k4 + 2.0 * d1d2 * xi) / 8.0 + 
                                  dxi * d1d2 / 9.0)))))))); 
        }
    //}
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }

    result__[ib] = sdata[0];
}

extern "C" double spline_inner_product_gpu_v2(int size__, double const* x__, double const* dx__, double const* f__, 
                                              double const* g__, double* d_buf__, double* h_buf__, int stream_id__)
{
    hipStream_t stream = (stream_id__ == -1) ? NULL : streams[stream_id__];

    dim3 grid_t(256);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    //double* d_result;
    //CALL_CUDA(hipMalloc, (&d_result, grid_b.x * sizeof(double)));

    spline_inner_product_gpu_kernel_v2 <<<grid_b, grid_t, grid_t.x * sizeof(double), stream>>>
    (
        size__,
        x__,
        dx__,
        f__,
        g__,
        d_buf__
    );

    //double* h_result = (double*)malloc(grid_b.x * sizeof(double));
    CALL_CUDA(hipMemcpyAsync, (h_buf__, d_buf__, grid_b.x * sizeof(double), hipMemcpyDeviceToHost, stream));
    CALL_CUDA(hipStreamSynchronize, (stream));
    
    //hipMemcpy(h_result, d_result, grid_b.x * sizeof(double), hipMemcpyDeviceToHost);
    //CALL_CUDA(hipFree, (d_result));

    double result = 0;
    for (int ib = 0; ib < grid_b.x; ib++) result += h_buf__[ib];
    //free(h_result);
    
    return result;
}



//==================================
// High-level functions and kernels
//==================================

template <typename T, typename U>
__device__ U spline_inner_product_gpu_function(int ld, int size, double* r_dr, T* s1_coefs, U* s2_coefs)
{
    int N = size / blockDim.x;
    if (size % blockDim.x != 0) N++;

    extern __shared__ char sdata_ptr[];
    U* sdata = (U*)&sdata_ptr[0];

    int a_offs = 0 * ld;
    int b_offs = 1 * ld;
    int c_offs = 2 * ld;
    int d_offs = 3 * ld;

    sdata[threadIdx.x] = 0;

    for (int n = 0; n < N; n++)
    {
        int i = n * blockDim.x + threadIdx.x;
        if (i < size - 1)
        {
            double x0 = r_dr[i];
            double dx = r_dr[ld + i];

            T a1 = s1_coefs[a_offs + i];
            T b1 = s1_coefs[b_offs + i];
            T c1 = s1_coefs[c_offs + i];
            T d1 = s1_coefs[d_offs + i];
            
            U a2 = s2_coefs[a_offs + i];
            U b2 = s2_coefs[b_offs + i];
            U c2 = s2_coefs[c_offs + i];
            U d2 = s2_coefs[d_offs + i];
                
            U a1a2 = a1 * a2;
            U d1d2 = d1 * d2;
                
            U k1 = d1 * b2 + c1 * c2 + b1 * d2;

            U k2 = d1 * a2 + c1 * b2 + b1 * c2 + a1 * d2;

            U k3 = c1 * a2 + b1 * b2 + a1 * c2;

            U k4 = d1 * c2 + c1 * d2;
            
            U k5 = b1 * a2 + a1 * b2;

            sdata[threadIdx.x] += dx * ((a1a2 * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2 + x0 * k5)) / 2.0 +
                                  dx * ((a1a2 + x0 * (2.0 * k5 + k3 * x0)) / 3.0 + 
                                  dx * ((k5 + x0 * (2.0 * k3 + k2 * x0)) / 4.0 +
                                  dx * ((k3 + x0 * (2.0 * k2 + k1 * x0)) / 5.0 + 
                                  dx * ((k2 + x0 * (2.0 * k1 + k4 * x0)) / 6.0 + 
                                  dx * ((k1 + x0 * (2.0 * k4 + d1d2 * x0)) / 7.0 + 
                                  dx * ((k4 + 2.0 * d1d2 * x0) / 8.0 + 
                                  dx * d1d2 / 9.0)))))))); 
        }
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }
    
    //if (threadIdx.x == 0) for (int i = 1; i < blockDim.x; i++) sdata[0] += sdata[i];

    return sdata[0];
}

template <> __device__ 
hipDoubleComplex spline_inner_product_gpu_function<double, hipDoubleComplex>(int ld, int size, double* r_dr, 
                                                                           double* s1_coefs, 
                                                                           hipDoubleComplex* s2_coefs)
{
    int N = size / blockDim.x;
    if (size % blockDim.x != 0) N++;

    extern __shared__ char sdata_ptr[];
    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];

    int a_offs = 0 * ld;
    int b_offs = 1 * ld;
    int c_offs = 2 * ld;
    int d_offs = 3 * ld;

    sdata[threadIdx.x] = make_hipDoubleComplex(0.0, 0.0);

    for (int n = 0; n < N; n++)
    {
        int i = n * blockDim.x + threadIdx.x;
        if (i < size - 1)
        {
            double x0 = r_dr[i];
            double dx = r_dr[ld + i];

            double a1 = s1_coefs[a_offs + i];
            double b1 = s1_coefs[b_offs + i];
            double c1 = s1_coefs[c_offs + i];
            double d1 = s1_coefs[d_offs + i];
            
            hipDoubleComplex a2 = s2_coefs[a_offs + i];
            hipDoubleComplex b2 = s2_coefs[b_offs + i];
            hipDoubleComplex c2 = s2_coefs[c_offs + i];
            hipDoubleComplex d2 = s2_coefs[d_offs + i];
                
            hipDoubleComplex a1a2 = make_hipDoubleComplex(a1 * a2.x, a1 * a2.y);
            hipDoubleComplex d1d2 = make_hipDoubleComplex(d1 * d2.x, d1 * d2.y);
                
            hipDoubleComplex k1 = make_hipDoubleComplex(d1 * b2.x + c1 * c2.x + b1 * d2.x, 
                                                      d1 * b2.y + c1 * c2.y + b1 * d2.y);

            hipDoubleComplex k2 = make_hipDoubleComplex(d1 * a2.x + c1 * b2.x + b1 * c2.x + a1 * d2.x, 
                                                      d1 * a2.y + c1 * b2.y + b1 * c2.y + a1 * d2.y);

            hipDoubleComplex k3 = make_hipDoubleComplex(c1 * a2.x + b1 * b2.x + a1 * c2.x, 
                                                      c1 * a2.y + b1 * b2.y + a1 * c2.y);

            hipDoubleComplex k4 = make_hipDoubleComplex(d1 * c2.x + c1 * d2.x, d1 * c2.y + c1 * d2.y);
            
            hipDoubleComplex k5 = make_hipDoubleComplex(b1 * a2.x + a1 * b2.x, b1 * a2.y + a1 * b2.y);

            hipDoubleComplex z = make_hipDoubleComplex(
                                  dx * ((a1a2.x * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2.x + x0 * k5.x)) / 2.0 +
                                  dx * ((a1a2.x + x0 * (2.0 * k5.x + k3.x * x0)) / 3.0 + 
                                  dx * ((k5.x + x0 * (2.0 * k3.x + k2.x * x0)) / 4.0 +
                                  dx * ((k3.x + x0 * (2.0 * k2.x + k1.x * x0)) / 5.0 + 
                                  dx * ((k2.x + x0 * (2.0 * k1.x + k4.x * x0)) / 6.0 + 
                                  dx * ((k1.x + x0 * (2.0 * k4.x + d1d2.x * x0)) / 7.0 + 
                                  dx * ((k4.x + 2.0 * d1d2.x * x0) / 8.0 + 
                                  dx * d1d2.x / 9.0)))))))),
                                  dx * ((a1a2.y * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2.y + x0 * k5.y)) / 2.0 +
                                  dx * ((a1a2.y + x0 * (2.0 * k5.y + k3.y * x0)) / 3.0 + 
                                  dx * ((k5.y + x0 * (2.0 * k3.y + k2.y * x0)) / 4.0 +
                                  dx * ((k3.y + x0 * (2.0 * k2.y + k1.y * x0)) / 5.0 + 
                                  dx * ((k2.y + x0 * (2.0 * k1.y + k4.y * x0)) / 6.0 + 
                                  dx * ((k1.y + x0 * (2.0 * k4.y + d1d2.y * x0)) / 7.0 + 
                                  dx * ((k4.y + 2.0 * d1d2.y * x0) / 8.0 + 
                                  dx * d1d2.y / 9.0)))))))));

            sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], z);
        }
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        __syncthreads();
    }
    
    //if (threadIdx.x == 0) for (int i = 1; i < blockDim.x; i++) sdata[0] = hipCadd(sdata[0], sdata[i]);

    return sdata[0];
}

template <typename T, typename U>
__global__ void spline_inner_product_gpu_kernel(int ld, int size, double* r_dr, T* s1_coefs, U* s2_coefs, U* result)
{
    result[0] = spline_inner_product_gpu_function(ld, size, r_dr, s1_coefs, s2_coefs);
}

template <typename T>
void spline_inner_product_gpu(int size, double* r_dr, T* s1_coefs, T* s2_coefs)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(1);

    T* d_result;
    hipMalloc(&d_result, 1 * sizeof(T));
    spline_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>(size, size, r_dr, s1_coefs, s2_coefs, d_result);

    T* h_result = (T*)malloc(1 * sizeof(T));
    hipMemcpy(h_result, d_result, 1 * sizeof(T), hipMemcpyDeviceToHost);

    printf("GPU result : %18.12f \n", h_result[0]);

    hipFree(d_result);
    free(h_result);
    
    //hipDoubleComplex* d_zresult;
    //hipMalloc(&d_zresult, 1 * sizeof(hipDoubleComplex));
    //
    //hipDoubleComplex* zs2;
    //hipMalloc(&zs2, size * 4 * sizeof(hipDoubleComplex));
    //
    //for (int i = 0; i < size * 4; i++) zs2[i] = make_hipDoubleComplex(s2_coefs[i], s2_coefs[i]);

    //spline_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>(size, size, r_dr, s1_coefs, zs2, d_zresult);

    //hipDoubleComplex* h_zresult = (hipDoubleComplex*)malloc(1 * sizeof(hipDoubleComplex));
    //hipMemcpy(h_zresult, d_zresult, 1 * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //printf("GPU result : %18.12f %18.12f\n", h_zresult[0].x, h_zresult[0].y);

    //hipFree(d_zresult);
    //free(h_zresult);
    //free(zs2);
}

template void spline_inner_product_gpu<double>(int size, double* r_dr, double* s1_coefs, double* s2_coefs);







// Input array dimensions:
//   sbessel_coefs(max_num_mt_points * 4, lmax_pw + 1, num_atom_types, num_gkvec_row);
//   lo_coefs(max_num_mt_points * 4, num_lo);
//   jlo(num_gkvec, num_lo);
__global__ void sbessel_lo_inner_product_gpu_kernel(int* kargs, int num_gkvec, int* l_by_ilo, int* iat_by_ilo, 
                                                    int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, 
                                                    double* lo_coefs, double* jlo)
{
    int num_atom_types = kargs[0];
    int max_nmtp = kargs[1];
    int lmax_pw = kargs[2];

    int igk = blockIdx.x;
    int ilo = blockIdx.y;

    int l = l_by_ilo[ilo];
    int iat = iat_by_ilo[ilo];
    int nmtp = nmtp_by_iat[iat];

    double* jl_ptr = &sbessel_coefs[array4D_offset(0, l, iat, igk, max_nmtp * 4, lmax_pw + 1, num_atom_types)];
    double* lo_ptr = &lo_coefs[array2D_offset(0, ilo, max_nmtp * 4)];
    double* r_dr_ptr = &r_dr[array2D_offset(0, iat, 2 * max_nmtp)];
    
    jlo[array2D_offset(igk, ilo, num_gkvec)] = 
        spline_inner_product_gpu_function(max_nmtp, nmtp, r_dr_ptr, jl_ptr, lo_ptr);
}


void sbessel_lo_inner_product_gpu(int* kargs, int num_gkvec, int num_lo, int* l_by_ilo, int* iat_by_ilo, 
                                  int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, double* lo_coefs, double* jlo)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_gkvec, num_lo);

    sbessel_lo_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>
        (kargs, num_gkvec, l_by_ilo, iat_by_ilo, nmtp_by_iat, r_dr, sbessel_coefs, lo_coefs, jlo);
}

// Compute <jl|V|lo>
// Input array dimensions:
//   vlo(max_num_mt_points * 4, lmmax_pw, num_lo_col)
//   jvlo(lmmax_pw, num_gkvec, num_lo)
__global__ void sbessel_vlo_inner_product_gpu_kernel(int* kargs, int num_gkvec, int* l_by_lm, int* iat_by_ilo, 
                                                     int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, 
                                                     hipDoubleComplex* vlo_coefs, hipDoubleComplex* jvlo)
{
    int num_atom_types = kargs[0];
    int max_nmtp = kargs[1];
    int lmax_pw = kargs[2];
    int lmmax_pw = kargs[3];

    int igk = blockIdx.x;
    int ilo = blockIdx.y;
    int lm = blockIdx.z;

    int l = l_by_lm[lm];
    int iat = iat_by_ilo[ilo];
    int nmtp = nmtp_by_iat[iat];
    
    double* jl_ptr = &sbessel_coefs[array4D_offset(0, l, iat, igk, max_nmtp * 4, lmax_pw + 1, num_atom_types)];
    hipDoubleComplex* vlo_ptr = &vlo_coefs[array3D_offset(0, lm, ilo, 4 * max_nmtp, lmmax_pw)];
    double* r_dr_ptr = &r_dr[array2D_offset(0, iat, 2 * max_nmtp)];
    
    jvlo[array3D_offset(lm, igk, ilo, lmmax_pw, num_gkvec)] = 
        spline_inner_product_gpu_function(max_nmtp, nmtp, r_dr_ptr, jl_ptr, vlo_ptr);
}

// Compute <jl|V|lo>
void sbessel_vlo_inner_product_gpu(int* kargs, int num_gkvec, int num_lo, int lmmax_pw, int* l_by_lm, int* iat_by_ilo, 
                                   int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, void* vlo_coefs, void* jvlo)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_gkvec, num_lo, lmmax_pw);

    sbessel_vlo_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>
        (kargs, num_gkvec, l_by_lm, iat_by_ilo, nmtp_by_iat, r_dr, sbessel_coefs, (hipDoubleComplex*)vlo_coefs, 
         (hipDoubleComplex*)jvlo);
}

__global__ void sbessel_vlm_inner_product_gpu_kernel(int* kargs, int* iat_by_ia, int* l_by_lm, int* nmtp_by_iat,
                                                     double* r_dr, double* sbessel_coefs, double* vlm_coefs, 
                                                     double* jvlm)
{
    int max_nmtp = kargs[1];
    int lmax_pot = kargs[2];
    int lmmax_pot = kargs[3];
    
    int lm = blockIdx.x;
    int ia = blockIdx.y;

    int iat = iat_by_ia[ia];
    int nmtp = nmtp_by_iat[ia];
    int l = l_by_lm[lm];

    double* jl_ptr = &sbessel_coefs[array3D_offset(0, l, iat, max_nmtp * 4, lmax_pot + 1)];
    double* vlm_ptr = &vlm_coefs[array3D_offset(0, lm, ia, max_nmtp * 4, lmmax_pot)];
    double* r_dr_ptr = &r_dr[array2D_offset(0, iat, 2 * max_nmtp)];

    jvlm[array2D_offset(lm, ia, lmmax_pot)] = 
        spline_inner_product_gpu_function(max_nmtp, nmtp, r_dr_ptr, jl_ptr, vlm_ptr);
}


void sbessel_vlm_inner_product_gpu(int* kargs, int lmmax_pot, int num_atoms, int* iat_by_ia, int* l_by_lm, 
                                   int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, double* vlm_coefs, 
                                   double* jvlm, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];
    dim3 threadsPerBlock(64);
    dim3 numBlocks(lmmax_pot, num_atoms);
    
    sbessel_vlm_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16, stream>>>
        (kargs, iat_by_ia, l_by_lm, nmtp_by_iat, r_dr, sbessel_coefs, vlm_coefs, jvlm);
}


//__global__ void add_band_density_gpu_kernel(int nmtp, int lmmax_rho, int max_nmtp, int max_num_gaunt, int* gaunt12_size, 
//                                            int* gaunt12_lm1_by_lm3, int* gaunt12_lm2_by_lm3, 
//                                            hipDoubleComplex* gaunt12_cg, hipDoubleComplex* fylm, double weight, 
//                                            int ia, double* dens)
//{
//    int ir = blockDim.x * blockIdx.x + threadIdx.x;
//    int lm = blockIdx.y;
//
//    int offs3 = array3D_offset(ir, lm, ia, max_nmtp, lmmax_rho);
//
//    if (ir < nmtp)
//    {
//        for (int k = 0; k < gaunt12_size[lm]; k++)
//        {
//            int offs = array2D_offset(k, lm, max_num_gaunt);
//            int lm1 = gaunt12_lm1_by_lm3[offs];
//            int lm2 = gaunt12_lm2_by_lm3[offs];
//            hipDoubleComplex cg = gaunt12_cg[offs];
//            
//            int offs1 = array2D_offset(ir, lm1, max_nmtp);
//            int offs2 = array2D_offset(ir, lm2, max_nmtp);
//
//            hipDoubleComplex z = hipCmul(hipConj(fylm[offs1]), fylm[offs2]);
//
//            dens[offs3] += weight * hipCreal(hipCmul(z, cg));
//        }
//    }
//}

__global__ void add_band_density_gpu_kernel(int lmmax_rho, int lmmax_wf, int max_nmtp, int* ia_by_ialoc, 
                                            int* iat_by_ia, int* nmtp_by_iat, int max_num_gaunt, 
                                            int* gaunt12_size, int* gaunt12_lm1_by_lm3, int* gaunt12_lm2_by_lm3, 
                                            hipDoubleComplex* gaunt12_cg, hipDoubleComplex* fylm, double weight, 
                                            double* dens)
{
    int lm = blockIdx.x;
    int ialoc = blockIdx.y;
    int ia = ia_by_ialoc[ialoc];
    int iat = iat_by_ia[ia];
    int nmtp = nmtp_by_iat[iat];

    int offs3 = array3D_offset(0, lm, ialoc, max_nmtp, lmmax_rho);

    int N = nmtp / blockDim.x;
    if (nmtp % blockDim.x != 0) N++;

    for (int k = 0; k < gaunt12_size[lm]; k++)
    {
        int offs = array2D_offset(k, lm, max_num_gaunt);

        int lm1 = gaunt12_lm1_by_lm3[offs];
        int lm2 = gaunt12_lm2_by_lm3[offs];
        hipDoubleComplex cg = gaunt12_cg[offs];
        
        int offs1 = array3D_offset(0, lm1, ia, max_nmtp, lmmax_wf);
        int offs2 = array3D_offset(0, lm2, ia, max_nmtp, lmmax_wf);
        
        for (int n = 0; n < N; n++)
        {
            int ir = n * blockDim.x + threadIdx.x;
            if (ir < nmtp)
            {
                hipDoubleComplex z = hipCmul(hipConj(fylm[offs1 + ir]), fylm[offs2 + ir]);

                dens[offs3 + ir] += weight * hipCreal(hipCmul(z, cg));
            }
        }
    }
}

void add_band_density_gpu(int lmmax_rho, int lmmax_wf, int max_nmtp, int num_atoms_loc, int* ia_by_ialoc, 
                          int* iat_by_ia, int* nmtp_by_iat, int max_num_gaunt, int* gaunt12_size, 
                          int* gaunt12_lm1_by_lm3, int* gaunt12_lm2_by_lm3, void* gaunt12_cg, void* fylm, 
                          double weight, double* dens)
{
    dim3 threadsPerBlock(128);
    dim3 numBlocks(lmmax_rho, num_atoms_loc);
    add_band_density_gpu_kernel<<<numBlocks, threadsPerBlock>>>
        (lmmax_rho, lmmax_wf, max_nmtp, ia_by_ialoc, iat_by_ia, nmtp_by_iat, max_num_gaunt, gaunt12_size, 
         gaunt12_lm1_by_lm3, gaunt12_lm2_by_lm3, (hipDoubleComplex*)gaunt12_cg, (hipDoubleComplex*)fylm, weight, dens);
}
    


__global__ void scale_matrix_columns_gpu_kernel
(
    int nrow,
    hipDoubleComplex* mtrx,
    double* a
)
{
    int icol = blockIdx.y;
    int irow = blockIdx.x * blockDim.x + threadIdx.x;
    if (irow < nrow) 
    {
        mtrx[array2D_offset(irow, icol, nrow)] =
            hipCmul(mtrx[array2D_offset(irow, icol, nrow)], make_hipDoubleComplex(a[icol], 0));
    }
}

// scale each column of the matrix by a column-dependent constant
extern "C" void scale_matrix_columns_gpu(int nrow,
                                        int ncol,
                                        hipDoubleComplex* mtrx,
                                        double* a)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow, grid_t.x), ncol);

    scale_matrix_columns_gpu_kernel <<<grid_b, grid_t>>>
    (
        nrow,
        mtrx,
        a
    );
}

__global__ void scale_matrix_rows_gpu_kernel
(
    int nrow,
    hipDoubleComplex* mtrx,
    double* v
)
{
    int icol = blockIdx.y;
    int irow = blockDim.x * blockIdx.x + threadIdx.x;
    if (irow < nrow) 
    {
        mtrx[array2D_offset(irow, icol, nrow)] = 
            hipCmul(mtrx[array2D_offset(irow, icol, nrow)], make_hipDoubleComplex(v[irow], 0));
    }
}

// scale each row of the matrix by a row-dependent constant
extern "C" void scale_matrix_rows_gpu(int nrow,
                                      int ncol,
                                      hipDoubleComplex* mtrx,
                                      double* v)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow, grid_t.x), ncol);

    scale_matrix_rows_gpu_kernel <<<grid_b, grid_t>>>
    (
        nrow,
        mtrx,
        v
    );
}

//== __global__ void update_it_density_matrix_0_gpu_kernel(int fft_size, 
//==                                                       int nfft_max, 
//==                                                       hipDoubleComplex* psi_it, 
//==                                                       double* wt,
//==                                                       double* it_density_matrix)
//== {
//==     int ir = blockIdx.x * blockDim.x + threadIdx.x;
//==     for (int i = 0; i < nfft_max; i++)
//==     {
//==         if (ir < fft_size)
//==         {
//==             hipDoubleComplex z = psi_it[array3D_offset(ir, i, 0, fft_size, nfft_max)];
//==             it_density_matrix[array2D_offset(ir, 0, fft_size)] += (z.x * z.x + z.y * z.y) * wt[i];
//==         }
//==     }
//== }

__global__ void update_it_density_matrix_1_gpu_kernel(int fft_size,
                                                      int nfft_max,
                                                      int ispn,
                                                      hipDoubleComplex const* psi_it,
                                                      double const* wt,
                                                      double* it_density_matrix)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < nfft_max; i++)
    {
        if (ir < fft_size)
        {
            hipDoubleComplex z = psi_it[array3D_offset(ir, i, ispn, fft_size, nfft_max)];
            it_density_matrix[array2D_offset(ir, ispn, fft_size)] += (z.x * z.x + z.y * z.y) * wt[i];
        }
    }
}


//extern "C" void update_it_density_matrix_gpu(int fft_size, 
//                                             int nfft_max, 
//                                             int num_spins, 
//                                             int num_mag_dims, 
//                                             hipDoubleComplex* psi_it, 
//                                             double* wt, 
//                                             double* it_density_matrix)
//{
//    CUDA_timer t("update_it_density_matrix_gpu");
//
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(fft_size, grid_t.x));
//
//    switch (num_mag_dims)
//    {
//        //== case 3:
//        //== {
//        //==     for (int ir = 0; ir < fft_->size(); ir++)
//        //==     {
//        //==         double_complex z = wfit(ir, 0) * conj(wfit(ir, 1)) * w;
//        //==         it_density_matrix(ir, 2) += 2.0 * real(z);
//        //==         it_density_matrix(ir, 3) -= 2.0 * imag(z);
//        //==     }
//        //== }
//        case 1:
//        {
//            update_it_density_matrix_1_gpu_kernel <<<grid_b, grid_t>>>
//            (
//                fft_size,
//                nfft_max,
//                psi_it,
//                wt,
//                it_density_matrix
//            );
//        }
//        case 0:
//        {
//            update_it_density_matrix_0_gpu_kernel <<<grid_b, grid_t>>>
//            (
//                fft_size,
//                nfft_max,
//                psi_it,
//                wt,
//                it_density_matrix
//            );
//        }
//    }
//}

extern "C" void update_it_density_matrix_1_gpu(int fft_size, 
                                               int ispin,
                                               hipDoubleComplex const* psi_it, 
                                               double const* wt, 
                                               double* it_density_matrix)
{
    CUDA_timer t("update_it_density_matrix_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(fft_size, grid_t.x));

    update_it_density_matrix_1_gpu_kernel <<<grid_b, grid_t>>>
    (
        fft_size,
        1,
        ispin,
        psi_it,
        wt,
        it_density_matrix
    );

//==     switch (num_mag_dims)
//==     {
//==         //== case 3:
//==         //== {
//==         //==     for (int ir = 0; ir < fft_->size(); ir++)
//==         //==     {
//==         //==         double_complex z = wfit(ir, 0) * conj(wfit(ir, 1)) * w;
//==         //==         it_density_matrix(ir, 2) += 2.0 * real(z);
//==         //==         it_density_matrix(ir, 3) -= 2.0 * imag(z);
//==         //==     }
//==         //== }
//==         case 1:
//==         {
//==             update_it_density_matrix_1_gpu_kernel <<<grid_b, grid_t>>>
//==             (
//==                 fft_size,
//==                 nfft_max,
//==                 psi_it,
//==                 wt,
//==                 it_density_matrix
//==             );
//==         }
//==         case 0:
//==         {
//==             update_it_density_matrix_0_gpu_kernel <<<grid_b, grid_t>>>
//==             (
//==                 fft_size,
//==                 nfft_max,
//==                 psi_it,
//==                 wt,
//==                 it_density_matrix
//==             );
//==         }
//==     }
}

inline __device__ uint32_t random(size_t seed)
{
    uint32_t h = 5381;

    return (h << (seed % 15)) + h;
}

__global__ void randomize_on_gpu_kernel
(
    double* ptr__,
    size_t size__
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size__) ptr__[i] = double(random(i)) / (1 << 31);
}

extern "C" void randomize_on_gpu(double* ptr, size_t size)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size, grid_t.x));

    randomize_on_gpu_kernel <<<grid_b, grid_t>>>
    (
        ptr,
        size
    );
}


