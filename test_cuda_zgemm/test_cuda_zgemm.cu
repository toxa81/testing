#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <sys/time.h>

inline double current_time()
{
    timeval t;
    gettimeofday(&t, NULL);
    return double(t.tv_sec) + double(t.tv_usec) / 1e6;
}

int main(int argn, char** argv)
{
    hipDoubleComplex* h_A;
    hipDoubleComplex* h_C;

    hipDoubleComplex* d_A;
    hipDoubleComplex* d_C;

    int N = 200;
    int n = 200;
    
    hipblasHandle_t handle;

    hipHostMalloc((void**)&h_A, N * n * sizeof(hipDoubleComplex), hipHostMallocDefault);
    hipHostMalloc((void**)&h_C, N * N * sizeof(hipDoubleComplex), hipHostMallocDefault);

    hipMalloc((void**)&d_A, N * n * sizeof(hipDoubleComplex));
    hipMalloc((void**)&d_C, N * N * sizeof(hipDoubleComplex));

    hipblasCreate(&handle);

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);

    for (int i = 0; i < N * n; i++) h_A[i] = make_hipDoubleComplex(double(rand()) / RAND_MAX, double(rand()) / RAND_MAX);
    for (int i = 0; i < N * N; i++) h_C[i] = make_hipDoubleComplex(0, 0);


    hipMemcpy(d_A, h_A, n * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, N * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    

    hipMemcpy(d_A, h_A, n * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    
    hipStream_t stream;
    hipblasGetStream(handle, &stream);


    double t1 = -current_time();
    for (int ia = 0; ia < 20; ia++)
    {

        hipblasStatus_t status = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, n, &alpha, 
                                            d_A, N, d_A, N, &alpha, d_C, N);

        if (status != HIPBLAS_STATUS_SUCCESS)
        {

            printf("failed to execute cublasZgemm\n");
            
            switch (status)
            {
                case HIPBLAS_STATUS_NOT_INITIALIZED:
                {
                    printf("the library was not initialized\n");
                    break;
                }
                case HIPBLAS_STATUS_INVALID_VALUE:
                {
                    printf("the parameters m,n,k<0\n");
                    break;
                }
                case HIPBLAS_STATUS_ARCH_MISMATCH:
                {
                    printf("the device does not support double-precision\n");
                    break;
                }
                case HIPBLAS_STATUS_EXECUTION_FAILED:
                {
                    printf("the function failed to launch on the GPU\n");
                    break;
                }
                default:
                {
                    printf("unknown error\n");
                    break;
                }
            }

            exit(-1);
        }
    }
    hipStreamSynchronize(stream);

    t1 += current_time();

    printf("performance: %12.6f GFlops\n", 20 * 8e-9 * N * N * n / t1);




    hipblasDestroy(handle);

    hipHostFree(h_A);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_C);
    
}
