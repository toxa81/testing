#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <execinfo.h>
#include <unistd.h>
#include <signal.h>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <sys/time.h>

#ifdef NDEBUG
#define CALL_CUDA(func__, args__)                                                                                  \
{                                                                                                                  \
    cudaError_t error = func__ args__;                                                                             \
    if (error != cudaSuccess)                                                                                      \
    {                                                                                                              \
        char nm[1024];                                                                                             \
        gethostname(nm, 1024);                                                                                     \
        printf("hostname: %s\n", nm);                                                                              \
        printf("Error in %s at line %i of file %s: %s\n", #func__, __LINE__, __FILE__, cudaGetErrorString(error)); \
    }                                                                                                              \
}
#else
#define CALL_CUDA(func__, args__)                                                                                  \
{                                                                                                                  \
    hipError_t error;                                                                                             \
    func__ args__;                                                                                                 \
    hipDeviceSynchronize();                                                                                       \
    error = hipGetLastError();                                                                                    \
    if (error != hipSuccess)                                                                                      \
    {                                                                                                              \
        char nm[1024];                                                                                             \
        gethostname(nm, 1024);                                                                                     \
        printf("hostname: %s\n", nm);                                                                              \
        printf("Error in %s at line %i of file %s: %s\n", #func__, __LINE__, __FILE__, hipGetErrorString(error)); \
    }                                                                                                              \
}
#endif

void cufft_error_message(hipfftResult result)
{
    switch (result)
    {
        case HIPFFT_INVALID_PLAN:
        {
            printf("CUFFT_INVALID_PLAN\n");
            break;
        }
        case HIPFFT_ALLOC_FAILED:
        {
            printf("CUFFT_ALLOC_FAILED\n");
            break;
        }
        case HIPFFT_INVALID_VALUE:
        {
            printf("CUFFT_INVALID_VALUE\n");
            break;
        }
        case HIPFFT_INTERNAL_ERROR:
        {
            printf("CUFFT_INTERNAL_ERROR\n");
            break;
        }
        case HIPFFT_SETUP_FAILED:
        {
            printf("CUFFT_SETUP_FAILED\n");
            break;
        }
        case HIPFFT_INVALID_SIZE:
        {
            printf("CUFFT_INVALID_SIZE\n");
            break;
        }
        default:
        {
            printf("unknown error code %i\n", result);
            break;
        }
    }
}

#define CALL_CUFFT(func__, args__)                                                  \
{                                                                                   \
    hipfftResult result;                                                             \
    if ((result = func__ args__) != HIPFFT_SUCCESS)                                  \
    {                                                                               \
        char nm[1024];                                                              \
        gethostname(nm, 1024);                                                      \
        printf("hostname: %s\n", nm);                                               \
        printf("Error in %s at line %i of file %s: ", #func__, __LINE__, __FILE__); \
        cufft_error_message(result);                                                \
        exit(-100);                                                                 \
    }                                                                               \
}

inline double current_time()
{
    timeval t;
    gettimeofday(&t, NULL);
    return double(t.tv_sec) + double(t.tv_usec) / 1e6;
}

int main(int argn, char** argv)
{
    hipStream_t stream1;
    hipStream_t stream2;

    CALL_CUDA(hipStreamCreate, (&stream1));
    CALL_CUDA(hipStreamCreate, (&stream2));

    hipfftHandle plan1;
    hipfftHandle plan2;

    CALL_CUFFT(hipfftCreate, (&plan1));
    CALL_CUFFT(hipfftCreate, (&plan2));

    CALL_CUFFT(hipfftSetAutoAllocation, (plan1, true));
    CALL_CUFFT(hipfftSetAutoAllocation, (plan2, true));

    int dims[] = {128, 128, 128};
    size_t work_size;
    CALL_CUFFT(hipfftMakePlanMany, (plan1, 3, dims, dims, 1, 1, dims, 1, 1, HIPFFT_Z2Z, 1, &work_size));
    CALL_CUFFT(hipfftMakePlanMany, (plan2, 3, dims, dims, 1, 1, dims, 1, 1, HIPFFT_Z2Z, 1, &work_size));

    CALL_CUFFT(hipfftSetStream, (plan1, stream1));
    CALL_CUFFT(hipfftSetStream, (plan2, stream2));
    
    size_t buf_size = dims[0] * dims[1] * dims[2] * sizeof(hipDoubleComplex);
    hipDoubleComplex* buf1;
    hipDoubleComplex* buf2;
    CALL_CUDA(hipMalloc, (&buf1, buf_size));
    CALL_CUDA(hipMalloc, (&buf2, buf_size));

    CALL_CUDA(hipMemset, (buf1, 0, buf_size));
    CALL_CUDA(hipMemset, (buf2, 0, buf_size));

    double t = -current_time();
    for (int i = 0; i < 200; i++)
    {
        CALL_CUFFT(hipfftExecZ2Z, (plan1, buf1, buf1, HIPFFT_FORWARD));
    }
    CALL_CUDA(hipStreamSynchronize, (stream1));
    t += current_time();
    printf("Execution time: %.4f sec.\n", t);
    
    t = -current_time();
    for (int i = 0; i < 100; i++)
    {
        CALL_CUFFT(hipfftExecZ2Z, (plan1, buf1, buf1, HIPFFT_FORWARD));
        CALL_CUFFT(hipfftExecZ2Z, (plan2, buf2, buf2, HIPFFT_FORWARD));
    }
    //for (int i = 0; i < 100; i++)
    //{
    //    CALL_CUFFT(cufftExecZ2Z, (plan2, buf2, buf2, CUFFT_FORWARD));
    //}

    CALL_CUDA(hipStreamSynchronize, (stream1));
    CALL_CUDA(hipStreamSynchronize, (stream2));

    t += current_time();
    printf("Execution time: %.4f sec.\n", t);

    CALL_CUDA(hipFree, (buf1));
    CALL_CUDA(hipFree, (buf2));

    CALL_CUFFT(hipfftDestroy, (plan1));
    CALL_CUFFT(hipfftDestroy, (plan2));

    CALL_CUDA(hipStreamDestroy, (stream1));
    CALL_CUDA(hipStreamDestroy, (stream2));

}
